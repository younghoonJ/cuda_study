#include <hip/hip_runtime.h>

#include <chrono>
#include <cstring>
#include <iostream>
#include <ostream>
#include <random>
#include <vector>

namespace yh {
// control
constexpr bool PRINT_ERR_LOG = false;

template<typename T>
size_t
getMemSize(size_t numData) {
    return sizeof(T) * numData;
}

template<typename T>
using DevicePtr = T *;

template<typename T>
DevicePtr<T>
deviceMallocInit(size_t num_data) {
    T *ptr_;
    auto err = hipMalloc(&ptr_, num_data * sizeof(T));
    if (PRINT_ERR_LOG) {
        printf("cudaMalloc::cudaMalloc: %s\n", hipGetErrorName(err));
    }
    err = hipMemset(ptr_, 0, num_data * sizeof(T));
    if (PRINT_ERR_LOG) {
        printf("cudaMalloc::cudaMemset: %s\n", hipGetErrorName(err));
    }
    return ptr_;
}

template<typename T>
void
deviceMemFree(DevicePtr<T> ptr) {
    auto err = hipFree(ptr);
    if (PRINT_ERR_LOG) {
        printf("cudaFree: %s\n", hipGetErrorName(err));
    }
}

template<typename T>
using HostPtr = T *;

template<typename T>
HostPtr<T>
hostMallocInit(size_t num_data) {
    T *arr = new T[num_data];
    memset(arr, 0, sizeof(T) * num_data);
    return arr;
}

template<typename T>
void
push(DevicePtr<T> dst, const HostPtr<T> src, size_t count) {
    const auto err =
        hipMemcpy(dst, src, sizeof(T) * count, hipMemcpyHostToDevice);
    if (PRINT_ERR_LOG) {
        // ReSharper disable once CppDFAUnreachableCode
        printf("push:cudaMemcpy: %s\n", hipGetErrorName(err));
    }
}

template<typename T>
inline void
pull(HostPtr<T> dst, const DevicePtr<T> src, size_t count) {
    const auto err =
        hipMemcpy(dst, src, sizeof(T) * count, hipMemcpyDeviceToHost);
    if (PRINT_ERR_LOG) {
        // ReSharper disable once CppDFAUnreachableCode
        printf("pull:cudaMemcpy: %s\n", hipGetErrorName(err));
    }
}

class NTimer {
    enum class TimerStatus {
        ON  = 0,
        OFF = 1,
    };
    using clock_ = std::chrono::high_resolution_clock;

    class Timer_ {
        TimerStatus status = TimerStatus::OFF;
        std::chrono::time_point<clock_> t_measure_start;
        std::chrono::duration<double> t_acc;
        const char *name_;


    public:
        explicit Timer_(const char *name) : name_(name) { reset(); }

        void reset() {
            status = TimerStatus::OFF;
            t_acc  = std::chrono::duration<double>::zero();
        }

        void tick() {
            if (status == TimerStatus::ON) return;

            status          = TimerStatus::ON;
            t_measure_start = clock_::now();
        }

        void tock() {
            if (status == TimerStatus::OFF) return;

            status = TimerStatus::OFF;
            t_acc += (clock_::now() - t_measure_start);
        }

        friend std::ostream &operator<<(std::ostream &os, const Timer_ &obj) {
            const auto t_mills =
                std::chrono::duration_cast<std::chrono::nanoseconds>(obj.t_acc)
                    .count() /
                1000000.0;
            return os << obj.name_ << ": " << t_mills << " ms";
        }
    };

    bool isOn_ = true;
    std::vector<Timer_> timers;

public:
    NTimer() = default;

    void reset() {
        for (auto &timer : timers)
            timer.reset();
    }

    /*
    Returns the index of the timer just added.
    */
    size_t addTimter(const char *timer_name) {
        timers.emplace_back(timer_name);
        return timers.size() - 1;
    }

    void tick(size_t timer_id) { timers.at(timer_id).tick(); }

    void tock(size_t timer_id) { timers.at(timer_id).tock(); }

    friend std::ostream &operator<<(std::ostream &os, const NTimer &obj) {
        os << "Timer Report[num_timers=" << obj.timers.size() << "]\n";
        for (auto i = 0; i < obj.timers.size(); ++i)
            os << "  " << i << ". " << obj.timers[i] << '\n';
        return os;
    };
};
}  // namespace yh

/*
Kernel when the matrix C is less then Max block size(1024)
A: m x k
B: k x n
C: m x n
*/
template<typename T>
__global__ void
matMul0(yh::DevicePtr<T> A, yh::DevicePtr<T> B, yh::DevicePtr<T> C, int m,
        int n, int k) {
    auto row = threadIdx.x;
    auto col = threadIdx.y;
    auto idx = row * n + col;

    C[idx] = 0;
    for (int i = 0; i < k; ++i) {
        C[idx] += (A[row * k + i] * B[i * n + col]);
    }
}

template<typename T>
__global__ void
matMul1(yh::DevicePtr<T> A, yh::DevicePtr<T> B, yh::DevicePtr<T> C, int m,
        int n, int k) {
    auto row = blockDim.x * blockIdx.x + threadIdx.x;
    auto col = blockDim.y * blockIdx.y + threadIdx.y;
    auto idx = row * n + col;

    if (row < m and col < n) {
        C[idx] = 0;
        for (int i = 0; i < k; ++i)
            C[idx] += A[row * k + i] * B[i * n + col];
    }
}

int
main() {
    // clang-format off
    yh::NTimer timer;
    const auto timer_kernel_host = timer.addTimter("Kernel Execution(Host)");
    const auto timer_kernel_device = timer.addTimter("Kernel Execution(Device)");
    const auto timer_memcp_device_host = timer.addTimter("MemCpy host to device ");
    const auto timer_memcp_host_device = timer.addTimter("MemCpy device to host ");

    // clang-format on

    using DType_ = int;

    int m = 1024, n = 2048, k = 1024;

    auto sizeA = yh::getMemSize<DType_>(m * k);
    auto sizeB = yh::getMemSize<DType_>(k * n);
    auto sizeC = yh::getMemSize<DType_>(m * n);

    auto A          = yh::hostMallocInit<DType_>(sizeA);
    auto B          = yh::hostMallocInit<DType_>(sizeB);
    auto host_ans   = yh::hostMallocInit<DType_>(sizeC);
    auto device_ans = yh::hostMallocInit<DType_>(sizeC);

    auto dA = yh::deviceMallocInit<DType_>(sizeA);
    auto dB = yh::deviceMallocInit<DType_>(sizeB);
    auto dC = yh::deviceMallocInit<DType_>(sizeC);

    std::random_device rd;
    std::mt19937 rng(rd());

    std::uniform_int_distribution<int> dis(0, 99);

    for (int i = 0; i < sizeA; i++)
        A[i] = dis(rng);
    for (int i = 0; i < sizeB; i++)
        B[i] = dis(rng);

    timer.tick(timer_kernel_host);
    //    device compute
    for (int row = 0; row < m; ++row) {
        for (int col = 0; col < n; ++col) {
            host_ans[row * n + col] = 0;
            for (int y = 0; y < k; ++y) {
                host_ans[row * n + col] += A[row * k + y] * B[y * n + col];
            }
        }
    }
    timer.tock(timer_kernel_host);

    timer.tick(timer_memcp_host_device);
    yh::push(dA, A, sizeA);
    yh::push(dB, B, sizeB);
    timer.tock(timer_memcp_host_device);

    dim3 grid, block;

    if (m * n <= 1024) {
        block = dim3(m, n);
        grid  = dim3(1, 1);

        timer.tick(timer_kernel_device);
        matMul0<<<grid, block>>>(dA, dB, dC, m, n, k);
        hipDeviceSynchronize();
        timer.tock(timer_kernel_device);

    } else {
        block = dim3(1, 512);
        grid  = dim3(ceil(float(m) / block.x), ceil(float(n) / block.y));

        timer.tick(timer_kernel_device);
        matMul1<<<grid, block>>>(dA, dB, dC, m, n, k);
        hipDeviceSynchronize();
        timer.tock(timer_kernel_device);
    }


    timer.tick(timer_memcp_device_host);
    yh::pull(device_ans, dC, sizeC);
    timer.tock(timer_memcp_device_host);

    yh::deviceMemFree(dA);
    yh::deviceMemFree(dB);
    yh::deviceMemFree(dC);

    size_t diff_count = 0;
    for (int row = 0; row < m; ++row) {
        for (int col = 0; col < n; ++col) {
            auto idx = row * n + col;
            if (host_ans[idx] != device_ans[idx]) {
                diff_count += 1;
                printf("diff: C[%d, %d], %d!=%d \n", row, col, host_ans[idx],
                       device_ans[idx]);
            }
        }
    }
    delete[] A;
    delete[] B;
    delete[] host_ans;
    delete[] device_ans;

    printf("A(%d,%d), B(%d,%d), C(%d,%d), grid:(%d, %d), block: (%d, %d)\n", m,
           k, k, n, m, n, grid.x, grid.y, block.x, block.y);


    std::cout << timer << std::endl;
    if (diff_count > 0) {
        printf("diff_count:%lld/%lld", diff_count, (m * n));
    }

    return 0;
}


#include <hip/hip_runtime.h>
#include <iostream>
// I don't know why i dont need to include cuda_runtime.h

void deviceMemInfo(const char *header = nullptr) {
    size_t free, total;
    hipMemGetInfo(&free, &total);

    if (header != nullptr) {
        printf("[%s] Device Mem: %lld/%lld bytes\n", header, free, total);
    } else {
        printf("Device Mem: %lld/%lld bytes\n", free, total);
    }
}

int main() {
    int *d_dataPtr;

    deviceMemInfo("start");

    auto err = hipMalloc(&d_dataPtr, sizeof(int) * 1024 * 1024);
    printf("cudaMalloc: %s\n", hipGetErrorName(err));
    deviceMemInfo("after hipMalloc");

    err = hipMemset(d_dataPtr, 2, sizeof(int) * 1024 * 1024);
    printf("cudaMemset: %s\n", hipGetErrorName(err));
    deviceMemInfo("after hipMemset");

    err = hipFree(d_dataPtr);
    printf("cudaFree: %s\n", hipGetErrorName(err));
    deviceMemInfo();

    return 0;
}

#include <iostream>
#include <hip/hip_runtime.h>

#include <memory>

__global__ void printDeviceData(const int *d_dataPtr) {
    printf("%d", d_dataPtr[threadIdx.x]);
}

__global__ void setData(int *d_dataPtr) {
    d_dataPtr[threadIdx.x] = 2;
}

int main() {
    int data[10];
    for (auto i = 0; i < 10; ++i) data[i] = 7;

    int *d_dataPtr;

    hipDeviceSynchronize();
    auto err = hipMalloc(&d_dataPtr, sizeof(int) * 10);
    err = hipMemset(d_dataPtr, 0, sizeof(int) * 10);
    printf("device data: ");
    printDeviceData<<<1, 10>>>(d_dataPtr);
    hipDeviceSynchronize();
    printf("\n");

    err = hipMemcpy(d_dataPtr, data, sizeof(int) * 10, hipMemcpyHostToDevice);
    printf("host to device: ");
    printDeviceData<<<1, 10>>>(d_dataPtr);
    hipDeviceSynchronize();
    printf("\n");

    setData<<<1, 10>>>(d_dataPtr);
    printf("device set data: ");
    printDeviceData<<<1, 10>>>(d_dataPtr);
    hipDeviceSynchronize();
    printf("\n");

    err = hipMemcpy(data, d_dataPtr, sizeof(int) * 10, hipMemcpyDeviceToHost);
    printf("device to host: ");
    for (auto i = 0; i < 10; ++i) printf("%d", data[i]);

    hipFree(d_dataPtr);
    return 0;
}

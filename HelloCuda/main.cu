#include <iostream>
#include <hip/hip_runtime.h>



__global__ void helloCuda() {
    printf("hello GPU!\n");
}

int main() {
    printf("Hello CPU!\n");
    helloCuda<<<1, 10>>>();
    hipDeviceSynchronize();

    return 0;
}
